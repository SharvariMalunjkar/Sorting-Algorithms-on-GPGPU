#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <cutil_inline.h>
#include<hip/hip_runtime.h>
//#define N 1000

int size;

// input array vlaues 
unsigned int *h_In =0;
unsigned int *h_Out = 0;
unsigned int *d_In = 0;
unsigned int *d_Out=0;


unsigned int timer_GPU =0;

void ParseArguments(int, char**);
void Cleanup(void);
/// Write kernel code here....///

__global__ void Selection_Sort(unsigned int *data_in,unsigned int *data_out, int left, int right)
{

    unsigned int tid = threadIdx.x;
 
   // for (int i = left ; i <= right ; ++i)
    for(int i =tid; i< (tid+right);i++)
    {
        unsigned min_val = data_in[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            unsigned val_j = data_in[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data_out[min_idx] = data_in[i];
            data_out[i] = min_val;
        }
    }
}








// Host code here...//
int main(int argc, char **argv)
{

  ParseArguments(argc,argv);
  int N =size;
  printf("sorting for number of elements %d\n",N);
  // unsigned int *h_data =0;
  // unsigned int *d_data = 0;
   
   // Allocate Host Memory
   h_In =(unsigned int*) malloc(N*sizeof(unsigned int));
   h_Out =(unsigned int*) malloc(N*sizeof(unsigned int));
   // Random number array in host memory
   
   for (int x = 0; x < N; ++x) {
                        h_In[x] = rand() % 100;
                        //printf("%d ", values[x];
   }
   // Device Memory allocation
   cutilSafeCall(hipMalloc((void **)&d_In, N * sizeof(unsigned int)));
   cutilSafeCall(hipMalloc((void **)&d_Out, N * sizeof(unsigned int)));
   // Copy data to GPU from CPU
   cutilSafeCall(hipMemcpy(d_In, h_In, N * sizeof(unsigned int), hipMemcpyHostToDevice));

   // Initializing the timer to zero cycles
   cutilCheckError(cutCreateTimer(&timer_GPU));

   // Start the timer
   cutilCheckError(cutStartTimer(timer_GPU));

   // call Kernel
   int left =0;
   int right = N-1;
   Selection_Sort<<<N,N>>>(d_In,d_Out,left,right);


  // Stop the timer
   cutilCheckError(cutStopTimer(timer_GPU));

   //Copy results back to CPU
   cutilSafeCall(hipMemcpy(d_Out, h_Out, N * sizeof(unsigned int), hipMemcpyHostToDevice)); 

   // checking for correctness of results

   printf("\nTesting results...\n");
                for (int x = 0; x < N - 1; x++) {
                        if (h_Out[x] > h_Out[x + 1]) {
                                printf("Sorting failed.\n");
                                break;
                        }
                        else
                                if (x == N - 2)
                                        printf("SORTING SUCCESSFUL\n");
                }
     // print the timer
   printf("GPU execution time is %f(ms)\n",cutGetTimerValue(timer_GPU));


     Cleanup();
     return 0;
   

}
void Cleanup(void)
{
// Free device memory
if (d_In)
   hipFree(d_In);
if(d_Out)
  hipFree(d_Out);

// free host memory

if (h_In)
   free(h_In);
if (h_Out)
   free(h_Out);

cutilCheckError(cutDeleteTimer(timer_GPU));
cutilSafeCall(hipDeviceReset());
exit(0);
}


void ParseArguments(int argc,char** argv)
{

   for(int i =0; i< argc;i++)
   {
      if (strcmp(argv[i],"--size")==0||strcmp(argv[i], "-size")==0){
         size = atoi (argv[i+1]);
         i = i+1;
       }
   }
}





