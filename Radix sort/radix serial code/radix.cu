#include "hip/hip_runtime.h"
#include<stdio.h>
# include<stdlib.h>
#include<cutil_inline.h>

//#define N 100

void ParseArguments(int, char**);

unsigned int timer_CPU =0;
int size;


int getMax(int arr[], int n)
{
    int mx = arr[0];
    for (int i = 1; i < n; i++)
        if (arr[i] > mx)
            mx = arr[i];
    return mx;
}
 
// A function to do counting sort of arr[] according to
// the digit represented by exp.
void countSort(int arr[], int n, int exp)
{
    int output[n]; // output array
    int i, count[10] = {0};
 
    // Store count of occurrences in count[]
    for (i = 0; i < n; i++)
        count[ (arr[i]/exp)%10 ]++;
 
    // Change count[i] so that count[i] now contains actual
    //  position of this digit in output[]
    for (i = 1; i < 10; i++)
        count[i] += count[i - 1];
 
    // Build the output array
    for (i = n - 1; i >= 0; i--)
    {
        output[count[ (arr[i]/exp)%10 ] - 1] = arr[i];
        count[ (arr[i]/exp)%10 ]--;
    }
 
    // Copy the output array to arr[], so that arr[] now
    // contains sorted numbers according to current digit
    for (i = 0; i < n; i++)
        arr[i] = output[i];
}
 
// The main function to that sorts arr[] of size n using 
// Radix Sort
void radixsort(int arr[], int n)
{
    // Find the maximum number to know number of digits
    int m = getMax(arr, n);
 
    // Do counting sort for every digit. Note that instead
    // of passing digit number, exp is passed. exp is 10^i
    // where i is current digit number
    for (int exp = 1; m/exp > 0; exp *= 10)
        countSort(arr, n, exp);
}
 
// A utility function to print an array
void print(int arr[], int n)
{
    for (int i = 0; i < n; i++)
        printf("Array is %d \n",arr[i]);
}
 
// Driver program to test above functions
int main(int argc, char** argv)
{
   ParseArguments(argc,argv);
   int N =size;
   printf("size is N:%d\n",N);
   // Initialize the timer to zero cycles
   cutilCheckError(cutCreateTimer(&timer_CPU));


    //int arr[] = {170, 45, 75, 90, 802, 24, 2, 66};
    int values[N];
     for (int x = 0; x < N; ++x) {
                        values[x] = rand() % 100;
                        //printf("%d ", values[x]);
                }
    int n = sizeof(values)/sizeof(values[0]);
   /* for (int i =0; i<n; i++){
    printf("given array is : %d\n",arr[i]);
    }*/

    // start the timer
    cutilCheckError(cutStartTimer(timer_CPU));
  
    radixsort(values, n);
    //print(arr, n);

    //stop timer
    cutilCheckError(cutStopTimer(timer_CPU));
     printf("\nTesting results...\n");
                for (int x = 0; x < N - 1; x++) {
                        if (values[x] > values[x + 1]) {
                                printf("Sorting failed.\n");
                                break;
                        }
                        else
                                if (x == N - 2)
                                        printf("SORTING SUCCESSFUL\n");
                }
    // print the timing
    printf("\n CPU timing is: %f(ms) for N (Number of elemenets getting sorted): %d\n",cutGetTimerValue(timer_CPU),N);

    return 0;
}

void ParseArguments(int argc,char** argv)
{

   for(int i =0; i< argc;i++)
   {
      if (strcmp(argv[i],"--size")==0||strcmp(argv[i], "-size")==0){
         size = atoi (argv[i+1]);
         i = i+1;
       }
   }
}
